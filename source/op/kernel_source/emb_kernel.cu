#include "hip/hip_runtime.h"
#include "emb_kernel.cuh"

namespace kernel {
    __global__ void emb_kernel_cu(float* in_weight, float* out, int32_t weight_dim) {
        const uint tid = threadIdx.x + blockDim.x * blockIdx.x;
        if (tid < weight_dim) {
            out[tid] = in_weight[tid];
        }
    }

    void emb_kernel_cuda(const mem::Tensor& input1, const mem::Tensor& weight, 
        const mem::Tensor& output, int32_t vocab_size) {
        const int32_t weight_dim = weight.get_dim(1);
        int32_t token = *input1.ptr<int32_t>();
        float* src_ptr = const_cast<float*>(weight.ptr<float>(token * weight_dim));
        float* dst_ptr = const_cast<float*>(output.ptr<float>());

        const uint block_size = 512;
        const uint grid_size = (weight_dim + block_size - 1) / block_size;
        emb_kernel_cu<<<grid_size, block_size>>>(src_ptr, dst_ptr, weight_dim);
    }
}